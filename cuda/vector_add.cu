#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel-функция, которая будет выполняться на GPU
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    // Получаем глобальный индекс потока
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Проверяем, чтобы не выйти за пределы массива
    // (важно, если размер массива не кратен числу потоков в блоке)
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 1000000; // Размер векторов
    size_t size = n * sizeof(float);

    // 1. Выделяем память на хосте (CPU)
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // Инициализируем векторы на хосте
    for (int i = 0; i < n; i++) {
        h_a[i] = (float)i;
        h_b[i] = (float)i * 2.0f;
    }

    // 2. Выделяем память на устройстве (GPU)
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // 3. Копируем данные с хоста на устройство
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // 4. Запускаем Kernel на GPU
    int threadsPerBlock = 256;
    // Рассчитываем количество блоков, чтобы покрыть все n элементов
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // <<<...>>> - это синтаксис для вызова Kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // 5. Копируем результат с устройства на хост
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // 6. Проверяем результат (опционально, но рекомендуется)
    for (int i = 0; i < 10; i++) {
        if (h_c[i] != (h_a[i] + h_b[i])) {
            printf("Ошибка на индексе %d!\n", i);
            break;
        }
    }
    printf("Вычисления на GPU завершены успешно.\n");
    printf("Пример результата: h_c[100] = %f\n", h_c[100]);

    // 7. Освобождаем память
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}